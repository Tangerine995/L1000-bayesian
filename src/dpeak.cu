// L1000 peak deconvolution based on Bayesian analysis
// 
// Copyright 2019 Tianhuan Lu
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//

#include <vector>

#include "hip/hip_runtime.h"


// maximum number of reads per bead color
#define CUDA_PROBPEAK_STRIDE 512

// scale parameter of the peak as a function of expression level x
#define PEAK_WIDTH_MODEL(x) (0.15f + 5.3f * exp2f(-0.75f * x))


__global__ void probpeak_kernel(const float bg1m, const float* values, const float* peakgrid, float* probpeak)
{
    int g = blockIdx.x;
    int i = threadIdx.x;

    float mean = peakgrid[g];
    float dev = PEAK_WIDTH_MODEL(mean);
    float inv_dev = 1.0f / dev;
    float prob = (values[i] - mean) * inv_dev;
    probpeak[g * CUDA_PROBPEAK_STRIDE + i] = bg1m * 3.30797f * powf(3.0f + prob * prob, -2.0f) * inv_dev;
}

__global__ void prob_kernel(const unsigned int num_values, unsigned int n_reduction,
    const float inv_num_values, const unsigned int grid_size,
    const float dp52_ratio, const float dp53_ratio,
    const float* probpeak, const float* probbg, float* intparams)
{
    int e52 = blockIdx.x;
    int e53 = blockIdx.y;
    int r = threadIdx.x;

    const float* dp52prob = probpeak + (e52 * CUDA_PROBPEAK_STRIDE);
    const float* dp53prob = probpeak + (e53 * CUDA_PROBPEAK_STRIDE);
    float* totalparams = intparams + (e52 * grid_size + e53) * 3;

    extern __shared__ float params[];

    float base = dp52_ratio * dp52prob[r] + dp53_ratio * dp53prob[r] + probbg[r];
    float diff = inv_num_values * (dp52prob[r] - dp53prob[r]) / base;

    params[r * 3 + 0] = __logf(base);
    params[r * 3 + 1] = diff;
    params[r * 3 + 2] = diff * diff;
    __syncthreads();

    if (r < int(num_values - n_reduction))
    {
        params[r * 3 + 0] += params[(r + n_reduction) * 3 + 0];
        params[r * 3 + 1] += params[(r + n_reduction) * 3 + 1];
        params[r * 3 + 2] += params[(r + n_reduction) * 3 + 2];
    }
    __syncthreads();
    n_reduction >>= 1;
    for (; n_reduction > 0; n_reduction >>= 1)
    {
        if (r < n_reduction)
        {
            params[r * 3 + 0] += params[(r + n_reduction) * 3 + 0];
            params[r * 3 + 1] += params[(r + n_reduction) * 3 + 1];
            params[r * 3 + 2] += params[(r + n_reduction) * 3 + 2];
        }
        __syncthreads();
    }
    if (r == 0)
    {
        totalparams[0] = params[0];
        totalparams[1] = params[1];
        totalparams[2] = params[2];
    }
}

__global__ void reduce_kernel(const float binomial_var, const float* intparams, float* likelihood)
{
    int e52 = blockIdx.x;
    int e53 = threadIdx.x;
    int stride = blockDim.x;

    const float* params_ptr = intparams + (e52 * stride + e53) * 3;
    float* likelihood_ptr = likelihood + (e52 * stride + e53);

    float a = params_ptr[0];
    float b = params_ptr[1];
    float c = params_ptr[2];
    *likelihood_ptr = a + (b * b * binomial_var) / (2.0f + 2.0f * c * binomial_var) - 0.5f * __logf(1.0f + c * binomial_var);
}

void cudadpeak_single(
    const float dp52_ratio, const float bg,
    const std::vector<float>& values, float* d_values,
    float* h_probpeak, float* d_probpeak,
    std::vector<float> probbg, float* h_probbg, float* d_probbg,
    const std::vector<float>& peakgrid, float* d_peakgrid,
    float* d_intparams, float* h_likelihood, float* d_likelihood)
{
    const size_t num_values = values.size();
    const size_t grid_size = peakgrid.size();

    if (num_values == 0)
    {
        std::fill_n(h_likelihood, grid_size * grid_size, 0.0f);
        return;
    }

    for (size_t i = 0; i < num_values; ++i)
        h_probbg[i] = probbg[i] * bg;
    hipMemcpy(d_probbg, h_probbg, sizeof(float) * num_values, hipMemcpyHostToDevice);
    hipMemcpy(d_values, values.data(), sizeof(float) * num_values, hipMemcpyHostToDevice);

    probpeak_kernel<<<grid_size, num_values>>>(1.0f - bg, d_values, d_peakgrid, d_probpeak);

    float inv_num_values = 1.0f / float(num_values);
    float binomial_var = float(num_values) * dp52_ratio * (1.0f - dp52_ratio);
    unsigned int n_reduction = 1;
    while (n_reduction * 2 < num_values)
        n_reduction *= 2;

    dim3 dim_grid(grid_size, grid_size);
    dim3 dim_block(num_values);
    size_t shared_memory_size = sizeof(float) * num_values * 3;
    prob_kernel<<<dim_grid, dim_block, shared_memory_size>>>(
        int(num_values), n_reduction, inv_num_values, int(grid_size),
        dp52_ratio, 1.0f - dp52_ratio, d_probpeak, d_probbg, d_intparams);

    std::vector<float> h_intparams(10000);
    hipMemcpy(h_intparams.data(), d_intparams, h_intparams.size() * 4, hipMemcpyDeviceToHost);

    reduce_kernel<<<grid_size, grid_size>>>(binomial_var, d_intparams, d_likelihood);
    hipMemcpy(h_likelihood, d_likelihood, sizeof(float) * grid_size * grid_size, hipMemcpyDeviceToHost);
}

std::vector<float> cudadpeak_batch(
    const std::vector<float>& dp52_ratio,                // ratio of dp52 beads
    const std::vector<float>& bg,                        // alpha_c in the paper
    const std::vector<std::vector<float>>& values_batch, // log2-FI in batches
    const std::vector<std::vector<float>>& probbg_batch, // PDF of all reads at each log2-FI
    const std::vector<float>& peakgrid)                  // grid points of peak locations
{
    const size_t batch_size = values_batch.size();
    const size_t grid_size = peakgrid.size();

    std::vector<float> probpeak(grid_size * CUDA_PROBPEAK_STRIDE);
    std::vector<float> probbg(CUDA_PROBPEAK_STRIDE);
    std::vector<float> likelihood_batch(batch_size * grid_size * grid_size);

    float* h_probpeak = probpeak.data();
    float* h_probbg = probbg.data();

    float* d_probpeak;
    float* d_values;
    float* d_probbg;
    float* d_peakgrid;
    float* d_intparams;
    float* d_likelihood;

    hipMalloc(&d_probpeak,   sizeof(float) * grid_size * CUDA_PROBPEAK_STRIDE);
    hipMalloc(&d_values,     sizeof(float) * CUDA_PROBPEAK_STRIDE);
    hipMalloc(&d_probbg,     sizeof(float) * CUDA_PROBPEAK_STRIDE);
    hipMalloc(&d_peakgrid,   sizeof(float) * grid_size);
    hipMalloc(&d_intparams,  sizeof(float) * grid_size * grid_size * 3);
    hipMalloc(&d_likelihood, sizeof(float) * grid_size * grid_size);

    hipMemcpy(d_peakgrid, peakgrid.data(), sizeof(float) * grid_size, hipMemcpyHostToDevice);

    for (size_t b = 0; b < batch_size; ++b)
    {
        float* h_likelihood = likelihood_batch.data() + b * grid_size * grid_size;
        cudadpeak_single(dp52_ratio[b], bg[b],
            values_batch[b], d_values,
            h_probpeak, d_probpeak,
            probbg_batch[b], h_probbg, d_probbg,
            peakgrid, d_peakgrid,
            d_intparams, h_likelihood, d_likelihood);
    }

    hipFree(d_likelihood);
    hipFree(d_intparams);
    hipFree(d_peakgrid);
    hipFree(d_probbg);
    hipFree(d_values);
    hipFree(d_probpeak);

    return likelihood_batch;
}

